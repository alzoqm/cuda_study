#include "hip/hip_runtime.h"
#include "tensor.h"
#include "matOper.h"
#include "matCal.h"
#include "Linear.h"
#include "init.h"

__host__ int main()
{
    // srand(time(NULL));
    // Tensor<float> tensor1({128, 1024, 1024});
    // Tensor<float> tensor2(1, {1024, 1024});
    // Tensor<float> tensor3({128, 1024, 1024});
    // //Tensor<float> tensor3_cpu({128, 1024, 1024});
    // Tensor<float> tensor4(3, {1024});
    // // clock_t cpu_start = clock();
    // // mat_mul(tensor1, tensor2, tensor3_cpu);
    // // clock_t cpu_end = clock();
    // // double cpu_time = (double)(cpu_end - cpu_start) / CLOCKS_PER_SEC;

    // size_t before_free, before_total;
    // hipMemGetInfo(&before_free, &before_total);
    // printf("Total GPU memory: %lu MB\nbefore Free GPU memory: %lu MB\n", before_total/1000000, before_free/1000000);
    
    // clock_t compile_start = clock();
    // tensor1.cuda();
    // tensor2.cuda();
    // tensor3.cuda();
    // tensor4.cuda();
    // clock_t compile_end = clock();
    // double compile_time = (double)(compile_end - compile_start) / CLOCKS_PER_SEC;

    // size_t free, total;
    // hipMemGetInfo(&free, &total);
    // printf("after Free GPU memory: %lu MB\n", free/1000000);
    // printf("diff before after memory: %lu MB\n\n", before_free/1000000- free/1000000);

    // clock_t gpu_start = clock();
    // mat_mul(tensor1, tensor2, tensor3);
    // matadd(tensor3, tensor4, tensor3);
    // clock_t gpu_end = clock();
    // double gpu_time = (double)(gpu_end - gpu_start) / CLOCKS_PER_SEC;

    // //printf("cpu_time: %f\n", cpu_time);
    // printf("GPU compile time: %f\n", compile_time);
    // printf("GPU time: %f\n", gpu_time);
    // printf("GPU sum time: %f\n", compile_time+gpu_time);
    // printf("\n");
    // //tensor3_cpu.print();
    // tensor3.print();

    size_t before_free, before_total;
    hipMemGetInfo(&before_free, &before_total);
    printf("Total GPU memory: %lu MB\nbefore Free GPU memory: %lu MB\n", before_total/1000000, before_free/1000000);
    clock_t create_start = clock();
    Tensor<float> *input = new Tensor<float>(2, {128, 1024, 1024});
    Linear<float> linear_1(1024, 1024, true);
    Tensor<float> *label = new Tensor<float>(2, {1024, 1024});
    clock_t create_end = clock();
    double create_time = (double)(create_end - create_start) / CLOCKS_PER_SEC;

    clock_t compile_start = clock();
    label->cuda();
    linear_1.cuda();
    input->cuda();
    clock_t compile_end = clock();
    double compile_time = (double)(compile_end - compile_start) / CLOCKS_PER_SEC;
    clock_t run_start = clock();
    Tensor<float>* out_ptr=nullptr;
    Tensor<float>* dx = nullptr;
    for(int i=0; i<1; i++)
    {
        if(out_ptr!=nullptr)
        {
            delete out_ptr;
        }
        out_ptr = linear_1.forward(input);
        if(dx!=nullptr)
        {
            delete dx;
        }
        dx = linear_1.backward(label);
    }
    clock_t run_end = clock();
    double run_time = (double)(run_end - run_start) / CLOCKS_PER_SEC;
    size_t free, total;
    hipMemGetInfo(&free, &total);
    printf("\nafter Free GPU memory: %lu MB\n", free/1000000);
    printf("diff before after memory: %lu MB\n\n", before_free/1000000- free/1000000);

    printf("create time: %f\n", create_time);
    printf("compile time: %f\n", compile_time);
    printf("run time: %f\n", run_time);
    printf("sum time: %f\n", run_time + compile_time + create_time);
    //out_ptr->print();
    dx->print();
    return 0;
}