#include "hip/hip_runtime.h"
#include "tensor.h"
#include "matOper.h"
#include "matCal.h"
#include "Linear.h"
#include "init.h"

__host__ int main()
{
    srand(time(NULL));
    Tensor<float> tensor1(2, {8, 128, 256});
    Tensor<float> tensor2(1, {256, 512});
    Tensor<float> tensor3({8, 128, 512});
    Tensor<float> tensor3_cpu({8, 128, 512});
    Tensor<float> tensor4(3, {512});
    clock_t cpu_start = clock();
    mat_mul(tensor1, tensor2, tensor3_cpu);
    clock_t cpu_end = clock();
    double cpu_time = (double)(cpu_end - cpu_start) / CLOCKS_PER_SEC;

    size_t before_free, before_total;
    hipMemGetInfo(&before_free, &before_total);
    printf("Total GPU memory: %lu MB\nbefore Free GPU memory: %lu MB\n", before_total/1000000, before_free/1000000);
    
    clock_t compile_start = clock();
    tensor1.cuda();
    tensor2.cuda();
    tensor3.cuda();
    tensor4.cuda();
    clock_t compile_end = clock();
    double compile_time = (double)(compile_end - compile_start) / CLOCKS_PER_SEC;

    size_t free, total;
    hipMemGetInfo(&free, &total);
    printf("after Free GPU memory: %lu MB\n", free/1000000);
    printf("diff before after memory: %lu MB\n\n", before_free/1000000- free/1000000);

    clock_t gpu_start = clock();
    mat_mul(tensor1, tensor2, tensor3);
    matadd(tensor3, tensor4, tensor3);
    clock_t gpu_end = clock();
    double gpu_time = (double)(gpu_end - gpu_start) / CLOCKS_PER_SEC;

    printf("cpu_time: %f\n", cpu_time);
    printf("GPU compile time: %f\n", compile_time);
    printf("GPU time: %f\n", gpu_time);
    printf("GPU sum time: %f\n", compile_time+gpu_time);
    printf("\n");
    tensor3_cpu.print();
    tensor3.print();

    Linear<float> linear_1(512, 1024, true);
    Tensor<float> input(2, {8, 128, 512});
    linear_1.cuda();
    input.cuda();
    clock_t gpu_start1 = clock();
    Tensor<float> output = linear_1.forward(input);
    clock_t gpu_end1 = clock();
    double gpu_time1 = (double)(gpu_end1 - gpu_start1) / CLOCKS_PER_SEC;
    printf("\nforward time: %f\n", gpu_time1);
    output.print();
    return 0;
}
